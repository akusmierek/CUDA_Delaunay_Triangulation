#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "delaunay.h"

#include <stdio.h>
#include "delaunayCuda.cuh"

// nvcc does not seem to like variadic macros, so we have to define
// one for each kernel parameter list:
#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t triangulateWithCuda();

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}


    ///////////////////
    // TRIANGULATION //
    ///////////////////
    std::vector<dt::Vector2<float>> points;
    points.push_back(dt::Vector2<float>{0, 2});
    points.push_back(dt::Vector2<float>{1, 0});
    points.push_back(dt::Vector2<float>{0, -2});
    points.push_back(dt::Vector2<float>{-10, 0});
    points.push_back(dt::Vector2<float>{2, 2});

    dt::Delaunay<float> triangulation;
    const std::vector<dt::Triangle<float>> triangles = triangulation.triangulate(points);
    const std::vector<dt::Edge<float>> edges = triangulation.getEdges();
    for (const auto& e : edges)
    {
        printf("edge from (%f, %f) to (%f, %f)\n", e.v->x, e.v->y, e.w->x, e.w->y);
    }

    printf("\n\n");

    std::vector<float2> cudaPoints;
    cudaPoints.push_back(make_float2(0, 2));
    cudaPoints.push_back(make_float2(1, 0));
    cudaPoints.push_back(make_float2(0, -2));
    cudaPoints.push_back(make_float2(-10, 0));
    cudaPoints.push_back(make_float2(2, 2));

    dtc::DelaunayCuda triangulationCuda;
    const std::vector<dtc::Triangle> trianglesCuda = triangulationCuda.triangulate(cudaPoints);
    const std::vector<dtc::Edge> edgesCuda = triangulationCuda.getEdges();
    for (const auto& e : edgesCuda)
    {
        printf("edge from (%f, %f) to (%f, %f)\n", e.v->x, e.v->y, e.w->x, e.w->y);
    }

    hipError_t triangulationCudaStatus = triangulateWithCuda();
    if (triangulationCudaStatus != hipSuccess)
    {
        fprintf(stderr, "triangulationWithCuda failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel KERNEL_ARGS2(1, size) (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

hipError_t triangulateWithCuda()
{
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    return cudaStatus;
}
